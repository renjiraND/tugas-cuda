#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <thrust/scan.h>

// fill random number to array
void rng(int* arr, int n) {
    int seed = 13516014; // NIM Renjira
    srand(seed);
    for (long i = 0; i < n; i++) {
        arr[i] = (int) rand();
    }
}

// get max number in array
__host__ int getMax(int *arr, int n) {
    int max = arr[0];
    for (int i = 1; i < n; i++) {
        if (arr[i] > max) {
            max = arr[i];
        }
    }
    return max;
}

__global__ void counting(int *arr, int *count, int n, int exp) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        int digit = (arr[i] / exp) % 10;
        atomicAdd(&count[digit], 1);
    }
    __syncthreads();
}

__host__ void parallelCountSort(int *arr, int n, int exp) {
    int output[n];

    int *d_arr;
    hipMalloc(&d_arr, n * sizeof (int));
    hipMemcpy(d_arr, arr, n * sizeof (int), hipMemcpyHostToDevice);
    int count[10] = {0};
    int *d_count;
    hipMalloc(&d_count, 10 * sizeof (int));
    hipMemcpy(d_count, count, 10 * sizeof (int), hipMemcpyHostToDevice);


    int grid_size = (n + 1023) / 1024;
    int thread_size = 1024;

    counting << <grid_size, thread_size>>>(d_arr, d_count, n, exp);

    hipMemcpy(count, d_count, 10 * sizeof (int), hipMemcpyDeviceToHost);

    thrust::inclusive_scan(count, count + 10, count);


    for (int i = n - 1; i >= 0; i--) {
        int digit = (arr[i] / exp) % 10;
        output[count[digit] - 1] = arr[i];
        count[digit]--;
    }

    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}

// parallel radix sort
__host__ void parallelRadixSort(int *arr, int n) {
    int m = getMax(arr, n);

    for (int exp = 1; m / exp > 0; exp *= 10) {
        parallelCountSort(arr, n, exp);
    }
}

// print array
__host__ void print(int *arr, int n) {
    printf("=======Array element:\n");
    for (int i = 0; i < n; i++) {
        printf("%d\n", arr[i]);
    }
    printf("=====================\n");
}

__host__ void serialCountSort(int *arr, int n, int exp) {
    int output[n];
    int count[10] = {0};

    for (int i = 0; i < n; i++) {
        int d = (arr[i] / exp) % 10;
        count[d]++;
    }

    for (int i = 1; i < 10; i++) {
        count[i] += count[i - 1];
    }

    for (int i = n - 1; i >= 0; i--) {
        int d = (arr[i] / exp) % 10;
        output[ count[d] - 1 ] = arr[i];
        count[d]--;
    }

    for (int i = 0; i < n; i++) {
        arr[i] = output[i];
    }
}

__host__ void serialRadixSort(int *arr, int n) {
    int m = getMax(arr, n);

    for (int exp = 1; m / exp > 0; exp *= 10) {
        serialCountSort(arr, n, exp);
    }
}

int main(int argc, char *argv[]) {
    hipEvent_t start, finish;
    float elapsedTime;
    if (argc < 2) {
        printf("usage: ./main N\n");
        exit(1);
    }
    int n = atoi(argv[1]);
    int *arr = (int *) malloc(n * sizeof (int));
    printf("Array size = %d\n", n);
    rng(arr, n);

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    serialRadixSort(arr, n);

    hipEventCreate(&finish);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);

    hipEventElapsedTime(&elapsedTime, start, finish);
    printf("Elapsed time for serial radix sort:  %f\n", elapsedTime);

    rng(arr, n);

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    parallelRadixSort(arr, n);

    hipEventCreate(&finish);
    hipEventRecord(finish, 0);
    hipEventSynchronize(finish);

    hipEventElapsedTime(&elapsedTime, start, finish);
    printf("Elapsed time for parallel radix sort:  %f\n", elapsedTime);
}